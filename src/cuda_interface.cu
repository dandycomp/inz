#include "hip/hip_runtime.h"
﻿#include "cuda_interface.cuh"

#include <iostream>
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>

using namespace cv;
using namespace std;

CudaInterface::CudaInterface(){
	m_img1_dev = NULL;
	m_img2_dev = NULL;

	m_glSum1_dev = NULL;
	m_glSum2_dev = NULL;

	m_glSqSum1_dev = NULL;
	m_glSqSum2_dev = NULL;
}

CudaInterface::~CudaInterface(){

	free(m_img1_host);
	free(m_img2_host);

	hipFree(m_img1_dev);
	hipFree(m_img2_dev);

	hipFree(m_glSum1_dev);
	hipFree(m_glSum2_dev);

	hipFree(m_glSqSum1_dev);
	hipFree(m_glSqSum2_dev);
}

void CudaInterface::setParameters(
	GlSumTbl& glimg1,
	GlSumTbl& glimg2){

	m_imgWidth = glimg1.m_img.cols;
	m_imgHeight = glimg1.m_img.rows;

	m_glWidth = glimg1.m_glSum.cols;
	m_glHeight = glimg1.m_glSum.rows;

	m_patch = glimg1.m_patch;
	m_subImg = glimg1.m_subImg;
	size_t numImageBytes = m_imgWidth* m_imgHeight * sizeof(int);
	size_t numGlSumBytes = m_glWidth *  m_glHeight * sizeof(int);

	m_img1_host = (int*)malloc(numImageBytes);//index format image
	m_img2_host = (int*)malloc(numImageBytes);

	m_glSum1_host = (int*)malloc(numGlSumBytes);
	int* m_glSum2_host = (int*)malloc(numGlSumBytes);

	m_glSqSum1_host = (int*)malloc(numGlSumBytes);
	int * m_glSqSum2_host = (int*)malloc(numGlSumBytes);

	//filling arrays of image data
	for (int i = 0; i < m_imgHeight; i++)
		for (int j = 0; j < m_imgWidth; j++)
		{
		int index = i*m_imgWidth + j;
		int tempPixelValue1 = glimg1.m_img.at<uchar>(i, j);
		int tempPixelValue2 = glimg2.m_img.at<uchar>(i, j);
		m_img1_host[index] = tempPixelValue1;
		m_img2_host[index] = tempPixelValue2;
		}

	//filling array of global sums and global squared sums
	for (int i = 0; i < m_glHeight; i++)
		for (int j = 0; j < m_glWidth; j++)
		{
		int index = i*m_glWidth + j;
		int tempGlSum1 = glimg1.m_glSum.at<float>(i, j);
		int tempGlSum2 = glimg2.m_glSum.at<float>(i, j);
		int tempGlSqSum1 = glimg1.m_glSqSum.at<float>(i, j);
		int tempGlSqSum2 = glimg2.m_glSqSum.at<float>(i, j);

		m_glSum1_host[index] = tempGlSum1;
		m_glSum2_host[index] = tempGlSum2;





		m_glSqSum1_host[index] = tempGlSqSum1;
		m_glSqSum2_host[index] = tempGlSqSum2;

		}


	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		this->~CudaInterface();
	}

	hipMalloc((void**)&m_img1_dev, numImageBytes);
	hipMalloc((void**)&m_img2_dev, numImageBytes);

	hipMalloc((void**)&m_glSum1_dev, numGlSumBytes);
	hipMalloc((void**)&m_glSum2_dev, numGlSumBytes);

	hipMalloc((void**)&m_glSqSum1_dev, numGlSumBytes);
	hipMalloc((void**)&m_glSqSum2_dev, numGlSumBytes);


	hipMemcpy(m_img1_dev, m_img1_host, numImageBytes, hipMemcpyHostToDevice);
	hipMemcpy(m_img2_dev, m_img2_host, numImageBytes, hipMemcpyHostToDevice);

	hipMemcpy(m_glSum1_dev, m_glSum1_host, numGlSumBytes, hipMemcpyHostToDevice);
	hipMemcpy(m_glSum2_dev, m_glSum2_host, numGlSumBytes, hipMemcpyHostToDevice);

	hipMemcpy(m_glSqSum1_dev, m_glSqSum1_host, numGlSumBytes, hipMemcpyHostToDevice);
	hipMemcpy(m_glSqSum2_dev, m_glSqSum2_host, numGlSumBytes, hipMemcpyHostToDevice);

	//free(m_glSum1_host);
	free(m_glSum2_host);
	//free(m_glSqSum1_host);
	free(m_glSqSum2_host);
}

void CudaInterface::array2Mat(float* arr){
	//zakladamy tylko poprawny rozmiar orazu
	int glWidth = IMAGE_WIDTH - PATCH_SIZE + 1;
	int glHeight = IMAGE_HEIGHT - PATCH_SIZE + 1;
	cv::Mat outpImg = cv::Mat::zeros(glHeight, glWidth, CV_8U);
	for (int i = 0; i < glHeight; i++)
		for (int j = 0; j < glWidth; j++)
		{
		int index = i*glWidth + j;
		float corr = arr[index];
		corr = 255 * (corr + 1.0) / 2.0;
		outpImg.at<uchar>(i, j) = (int)corr;
		}
	cv::imwrite("cudaOutp.jpg", outpImg);
}
void CudaInterface::run(){
	this->correlate();
	//getBestCorrFromArea_fast(20, 20);
}

void CudaInterface::correlate()
{
hipError_t start = hipProfilerStart();

int glWidth = IMAGE_WIDTH - PATCH_SIZE + 1;
int glHeight = IMAGE_HEIGHT - PATCH_SIZE + 1;
float* corrMat = new float[glWidth*glHeight];

int finishedPercent = 0;

for (int col = 0; col < glWidth; col++)
	for (int row = 0; row < glHeight; row++)
	{
	int index = row*glWidth + col;
	//corrMat[index] = getBestCorrFromArea(col + PATCH_SIZE / 2, row + PATCH_SIZE / 2);
	//corrMat[index] = simpleGetBestCorrFromArea(col + PATCH_SIZE / 2, row + PATCH_SIZE / 2);
	corrMat[index] = getBestCorrFromArea_fast(col + PATCH_SIZE / 2, row + PATCH_SIZE / 2);
	//cin.get();

	int actual = (col*glHeight + row) * 100
		/ (glWidth * glHeight);

	if (actual > finishedPercent)
	{
		finishedPercent = actual;
		cout << finishedPercent << " % " << endl;
	}
	}

hipError_t stop = hipProfilerStop();
array2Mat(corrMat);
}

float CudaInterface::getBestCorrFromArea_fast(int x, int y){

	// counting parameters of central patch of first image to simplfy and 
	// optimize algorithm. We need:
	//patch*patch array of pixel intensity values
	//medium value of this pixels
	//standart deviation of this patch

	int* im1_patch = new int[PATCH_SIZE*PATCH_SIZE];
	for (int row = 0; row < PATCH_SIZE; row++)
		for (int col = 0; col < PATCH_SIZE; col++)
		{
		int x_ = x - PATCH_SIZE / 2 + col;
		int y_ = y - PATCH_SIZE / 2 + row;

		int index = y_ * IMAGE_WIDTH + x_;
		int temp = m_img1_host[index];
		im1_patch[row * PATCH_SIZE + col] = temp;
		}

	int glImageWidth = IMAGE_WIDTH - PATCH_SIZE + 1;
	int index = x - PATCH_SIZE / 2 + (y - PATCH_SIZE / 2) * glImageWidth;
	int im1_patch_glSum = m_glSum1_host[index];
	int im1_patch_glSqSum = m_glSqSum1_host[index];

	dim3 threads(SUB_IMG, SUB_IMG);
	dim3 blocks(1, 1);

	int* im1_patch_device = NULL;
	hipMalloc((void**)&im1_patch_device, PATCH_SIZE*PATCH_SIZE*sizeof(int));
	hipMemcpy(im1_patch_device, im1_patch, PATCH_SIZE*PATCH_SIZE*sizeof(int), hipMemcpyHostToDevice);

	int area_side = SUB_IMG - PATCH_SIZE + 1;
	float* dev_corr = NULL;
	hipMalloc((void**)&dev_corr, area_side*area_side*sizeof(float));

	cudaGetBestCorrelate_fast << <blocks, threads >> >(m_img2_dev, m_glSum2_dev, m_glSqSum2_dev,
		im1_patch_device, x, y,
		im1_patch_glSum, im1_patch_glSqSum, dev_corr);

	float* host_corr =  new float [area_side*area_side];
	hipMemcpy(host_corr, dev_corr, area_side*area_side*sizeof(float), hipMemcpyDeviceToHost);

	float max_corr = -1;
	for(int i = 0; i < area_side*area_side; i++)
	{
		max_corr = host_corr[i] > max_corr? host_corr[i] : max_corr;
		//cout << " Corr = " << host_corr[i] << endl;
	}
	//cout << " Max correlation = " << max_corr << endl;
	return max_corr;
}


float CudaInterface::getBestCorrFromArea(int x, int y){

	int area_side = SUB_IMG - PATCH_SIZE + 1; // one side of results area matrix
	int area_size = area_side*area_side;// size of area

	dim3 threads(PATCH_SIZE, PATCH_SIZE);
	dim3 blocks(area_side, area_side);

	int *host_P_output = new int[area_size];// size of area around pixel
	for (int i = 0; i < area_size; i++)
		host_P_output[i] = 0.0;

	int *dev_P_output;
	hipMalloc((void**)&dev_P_output, area_size*sizeof(int));
	hipMemcpy(dev_P_output, host_P_output, area_size*sizeof(int), hipMemcpyHostToDevice);

	cudaGetAll_P_FromArea << <blocks, threads >> >(m_img1_dev, m_img2_dev,
		x, y, dev_P_output);

	hipDeviceSynchronize();

	//hipMemcpy(host_P_output, dev_P_output, area_size*sizeof(int), hipMemcpyDeviceToHost);
	//for (int i = 0; i < area_size; i++)
	//	cout << " p = " << host_P_output[i] << endl;

	//cin.get();



	float *host_output = new float[area_size];// size of area around pixel
	for (int i = 0; i < area_size; i++)
		host_output[i] = 0.0;

	float *dev_outp;
	hipMalloc((void**)&dev_outp, area_size*sizeof(float));
	hipMemcpy(dev_outp, host_output, area_size*sizeof(float), hipMemcpyHostToDevice);


	dim3 threads_2(area_side, area_side);
	cudaGetBestCorrelateFromArea << < 1, threads_2 >> >(dev_P_output, m_glSum1_dev,
		m_glSum2_dev, m_glSqSum1_dev, m_glSqSum2_dev,
		x, y, dev_outp);

	hipDeviceSynchronize();
	hipMemcpy(host_output, dev_outp, area_size*sizeof(float), hipMemcpyDeviceToHost);

	float max_cc = -1;
	for (int i = 0; i < area_size; i++){
		//cout << " host_output[i] = " << host_output[i] << endl;
		max_cc = host_output[i] > max_cc ? host_output[i] : max_cc;
	}

	hipFree(dev_P_output);
	hipFree(dev_outp);
	return max_cc;
}

float CudaInterface::simpleGetBestCorrFromArea(int x, int y){
	
	// counting parameters of central patch of first image to simplfy and 
	// optimize algorithm. We need:
	//patch*patch array of pixel intensity values
	//medium value of this pixels
	//standart deviation of this patch

	int* im1_patch = new int[PATCH_SIZE*PATCH_SIZE];
	int sum = 0;
	for (int row = 0; row < PATCH_SIZE; row++)
		for (int col = 0; col < PATCH_SIZE; col++)
		{
		int x_ = x - PATCH_SIZE / 2 + col;
		int y_ = y - PATCH_SIZE / 2 + row;

		int index = y_ * IMAGE_WIDTH + x_;
		int temp = m_img1_host[index];
		im1_patch[row * PATCH_SIZE + col] = temp;
		sum += temp;
		}

	int medium1 = (float)sum / ((float)PATCH_SIZE*(float)PATCH_SIZE);
	float stDev1 = 0;
	for (int i = 0; i < PATCH_SIZE*PATCH_SIZE; i++)
		stDev1 += pow((float)im1_patch[i] - medium1,2);

	int area_side = SUB_IMG - PATCH_SIZE + 1; // one side of results area matrix
	dim3 threads_(SUB_IMG, SUB_IMG);

	dim3 blocks_(1, 1);
	int* im1_patch_device = NULL;
	hipMalloc((void**)&im1_patch_device, PATCH_SIZE*PATCH_SIZE*sizeof(int));
	hipMemcpy(im1_patch_device, im1_patch, PATCH_SIZE*PATCH_SIZE*sizeof(int), hipMemcpyHostToDevice);

	float* dev_corr = NULL;
	hipMalloc((void**)&dev_corr, area_side*area_side*sizeof(float));


	cudaSimpleGetBestCorrelate<<<blocks_,threads_>>>(m_img2_dev,x,y,
		im1_patch_device, medium1, stDev1, dev_corr);

	float* host_corr =  new float [area_side*area_side];
	hipMemcpy(host_corr, dev_corr, area_side*area_side*sizeof(float), hipMemcpyDeviceToHost);
	
	float max_corr = -1;
	for(int i = 0; i < area_side*area_side; i++)
	{
		max_corr = host_corr[i] > max_corr? host_corr[i] : max_corr;
		//cout << " Corr = " << host_corr[i] << endl;
	}

	//cout << " Max correlation = " << max_corr << endl;

	return max_corr;

#if 0





	dim3 threads(PATCH_SIZE, PATCH_SIZE);
	dim3 blocks(area_side, area_side);

	int *host_P_output = new int[area_size];// size of area around pixel
	for (int i = 0; i < area_size; i++)
		host_P_output[i] = 0.0;

	int *dev_P_output;
	hipMalloc((void**)&dev_P_output, area_size*sizeof(int));
	hipMemcpy(dev_P_output, host_P_output, area_size*sizeof(int), hipMemcpyHostToDevice);

	cudaGetAll_P_FromArea << <blocks, threads >> >(m_img1_dev, m_img2_dev,
		x, y, dev_P_output);

	hipDeviceSynchronize();

	//hipMemcpy(host_P_output, dev_P_output, area_size*sizeof(int), hipMemcpyDeviceToHost);
	//for (int i = 0; i < area_size; i++)
	//	cout << " p = " << host_P_output[i] << endl;

	//cin.get();



	float *host_output = new float[area_size];// size of area around pixel
	for (int i = 0; i < area_size; i++)
		host_output[i] = 0.0;

	float *dev_outp;
	hipMalloc((void**)&dev_outp, area_size*sizeof(float));
	hipMemcpy(dev_outp, host_output, area_size*sizeof(float), hipMemcpyHostToDevice);


	dim3 threads_2(area_side, area_side);
	cudaGetBestCorrelateFromArea << < 1, threads_2 >> >(dev_P_output, m_glSum1_dev,
		m_glSum2_dev, m_glSqSum1_dev, m_glSqSum2_dev,
		x, y, dev_outp);

	hipDeviceSynchronize();
	hipMemcpy(host_output, dev_outp, area_size*sizeof(float), hipMemcpyDeviceToHost);

	float max_cc = -1;
	for (int i = 0; i < area_size; i++){
		//cout << " host_output[i] = " << host_output[i] << endl;
		max_cc = host_output[i] > max_cc ? host_output[i] : max_cc;
	}

	hipFree(dev_P_output);
	hipFree(dev_outp);
	return max_cc;
#endif
}

void CudaInterface::simpleCorrelate()
{
	hipError_t start = hipProfilerStart();
	int outpWidth = IMAGE_WIDTH - PATCH_SIZE + 1;
	int outpHeight = IMAGE_HEIGHT - PATCH_SIZE + 1;
	float* corrMat = new float[outpWidth*outpHeight];

	int finishedPercent = 0;

	for (int col = 0; col < outpWidth; col++)
		for (int row = 0; row < outpHeight; row++)
		{
		int index = row*outpWidth + col;
			
		//corrMat[index] = getBestCorrFromArea(col + PATCH_SIZE / 2, row + PATCH_SIZE / 2);

		int actual = (col*outpHeight + row) * 100
			/ (outpWidth * outpHeight);

		if (actual > finishedPercent)
		{
			finishedPercent = actual;
			cout << finishedPercent << " % " << endl;
		}
		}

	hipError_t stop = hipProfilerStop();
	array2Mat(corrMat);

}


void CudaInterface::deviceInfo(){
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	printf("There are %d CUDA devices.\n", devCount);

	hipDeviceProp_t devProp;
	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&devProp, i);
		printf("Major revision number:         %d\n", devProp.major);
		printf("Minor revision number:         %d\n", devProp.minor);
		printf("Name:                          %s\n", devProp.name);
		printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
		printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", devProp.regsPerBlock);
		printf("Warp size:                     %d\n", devProp.warpSize);
		printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
		printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
		printf("Clock rate:                    %d\n", devProp.clockRate);
		printf("Total constant memory:         %lu\n", devProp.totalConstMem);
		printf("Texture alignment:             %lu\n", devProp.textureAlignment);
		printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
		printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
		printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	}
}


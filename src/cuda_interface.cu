#include "hip/hip_runtime.h"
﻿#include "cuda_interface.cuh"

using namespace cv;
using namespace std;


CudaInterface::CudaInterface(){
	m_img1_dev = NULL;
	m_img2_dev = NULL;
}

CudaInterface::~CudaInterface(){
	hipFree(m_img1_dev);
	hipFree(m_img2_dev);
}

void CudaInterface::setParameters(
	Mat& img1, Mat& img2){

	int img_width = img1.cols;
	int img_height = img1.rows;

	int* img1_host = new int[img_height* img_width];
	int* img2_host = new int[img_height* img_width];

	//filling arrays of image data
	for (int i = 0; i < img_height; i++)
		for (int j = 0; j < img_width; j++)
		{
		int index = i*img_width + j;
		int tempPixelValue1 = img1.at<uchar>(i, j);
		int tempPixelValue2 = img2.at<uchar>(i, j);
		img1_host[index] = tempPixelValue1;
		img2_host[index] = tempPixelValue2;
		}

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		this->~CudaInterface();
	}

	size_t numImageBytes = img_height* img_width * sizeof(int);
	//memory allocation on device
	hipMalloc((void**)&m_img1_dev, numImageBytes);
	hipMalloc((void**)&m_img2_dev, numImageBytes);


	//copying data to device memory
	hipMemcpy(m_img1_dev, img1_host, numImageBytes, hipMemcpyHostToDevice);
	hipMemcpy(m_img2_dev, img2_host, numImageBytes, hipMemcpyHostToDevice);

	delete[] img1_host;
	delete[] img2_host;
}

void CudaInterface::fastCudaCorrelation(){
	time_t start, stop;
	double czas;
	start = clock();
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	int glWidth = IMAGE_WIDTH - SUB_IMG + 1;
	int glHeight = IMAGE_HEIGHT - SUB_IMG + 1;
	const int block_side = SUB_IMG - PATCH_SIZE + 1;

	float* corrMat_dev;
	//memory allocation on device
	cudaStatus = hipMalloc((void**)&corrMat_dev, glWidth*glHeight*block_side*block_side*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of corrMat_dev failed! Can't allocate memory \n ");
		return;
	}

	dim3 blocks(IMAGE_WIDTH / SUB_IMG + 1, IMAGE_HEIGHT / SUB_IMG + 1);
	dim3 threads(SUB_IMG, SUB_IMG);

	int nr_of_iter = SUB_IMG * SUB_IMG;
	int finishedPercent = 0;

	int dist = SUB_IMG / 2 + 1; // starting distortion of first reference image patch

	
	for (int row = 0; row < SUB_IMG; row++)
		for (int col = 0; col < SUB_IMG; col++){

		//int actual = (row*SUB_IMG + col) * 100
		//	/ (SUB_IMG*SUB_IMG);

		//if (actual > finishedPercent)
		//{
		//	finishedPercent = actual;
		//	cout << finishedPercent << " % " << endl;
		//}
		
		cudaFastCorrelation << <blocks, threads >> >(m_img1_dev, m_img2_dev, col+dist, row + dist, corrMat_dev);
		hipDeviceSynchronize();
		
		}

	
#if 1
	//start = clock();
	float* final_corrMat_dev;
	//memory allocation on device
	cudaStatus = hipMalloc((void**)&final_corrMat_dev, glWidth*glHeight*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of final_corrMat_dev failed! Can't allocate memory \n ");
		return;
	}
	

	int* posMat_dev;
	//memory allocation on device
	cudaStatus = hipMalloc((void**)&posMat_dev, glWidth*glHeight*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of posMat_dev failed! Can't allocate memory \n ");
		return;
	}
	
	int grid_size = 32;
	dim3 blocksToGetMax(glWidth/ grid_size + 1, glHeight/ grid_size + 1);
	dim3 threadsToGetMax(grid_size, grid_size);

	cudaGetMaxValues << <blocksToGetMax, threadsToGetMax >> >(corrMat_dev, final_corrMat_dev, posMat_dev);
	//cudaGetMaxValues << <blocksToGetMax, threadsToGetMax >> >(corrMat_dev, final_corrMat_dev);
	hipDeviceSynchronize();


	stop = clock();
	float* final_corrMat_host = new float[glWidth*glHeight];
	cudaStatus = hipMemcpy(final_corrMat_host, final_corrMat_dev,
		glWidth*glHeight*sizeof(float),
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! Can't copy final_corrMat_dev from device memory to host\n ");\
		cout << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	

	int* posMat_host = new int [glWidth*glHeight];
	cudaStatus = hipMemcpy(posMat_host, posMat_dev,
		glWidth*glHeight*sizeof(int),
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! Can't copy posMat from device memory to host\n ");
		cout << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	
	
	vector<vector< OutpStr>> outpVector;
	outpVector.clear();
	for(int i = 0; i < glHeight; i++)
	{
		vector< OutpStr> colResults;
		colResults.clear();
		for (int j = 0; j < glWidth; j++)
		{
			int startPosition = j + i*glWidth;

			Point ref = Point(j, i);
			int position = posMat_host[startPosition];
			int x_dist = position - ((int)(position/block_side))*block_side - block_side / 2;
			int y_dist = position/block_side - block_side / 2;

 
			//cout << "Position: " << position << " position%block_side: " << position%block_side << " block_side / 2 " << block_side / 2 << endl;
			//cout << "Distortions " << x_dist << " " << y_dist << endl;
			
			Point deform = Point(ref.x + x_dist, ref.y + y_dist);
			//cout << "Point ref = " << ref << " point deform = " << deform << endl;
			OutpStr tempStructure(ref, deform, final_corrMat_host[startPosition]);
			colResults.push_back(tempStructure);
			//cin.get();
		}
		outpVector.push_back(colResults);
	}
#endif

#if 0
	//start = clock();
	float* corrMat_host = new float[glWidth*glHeight*block_side*block_side];
	cudaStatus = hipMemcpy(corrMat_host, corrMat_dev,
		block_side*block_side*glWidth*glHeight*sizeof(float),
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! Can't copy final_corrMat_dev from device memory to host\n ");
		cout << hipGetErrorString(cudaStatus) << endl;
		this->~CudaInterface();
	}

	vector<vector< OutpStr>> outpVector;
	for(int i = 0; i < glHeight; i++)
	{
		vector< OutpStr> colResults;
		for (int j = 0; j < glWidth; j++)
		{
			float bestCorr = -1;
			Point pRecursive = Point(j, i);
			Point pDeformable = Point(0, 0);
			int startPosition = (j + i*glWidth)*block_side*block_side;

			int index_of_medium = startPosition + block_side / 2 + (block_side / 2 )*block_side;
			if (corrMat_host[index_of_medium] > 0.99)
			{
				OutpStr tempStructure(pRecursive, pRecursive, bestCorr);
				colResults.push_back(tempStructure);
			}
			else
			{
			for (int k = 0; k < block_side; k++)
			{
				for (int r = 0; r < block_side; r++)
				{
					int index = r + k * block_side + startPosition;
					float correlacja = corrMat_host[index];
					
					if (correlacja > bestCorr)
					{
						bestCorr = correlacja;
						pDeformable = Point(pRecursive.x + r - block_side / 2,
											pRecursive.y + k - block_side / 2);
					}
				}
			}
			
			OutpStr tempStructure(pRecursive, pDeformable, bestCorr);
			colResults.push_back(tempStructure);
			}
		}
		outpVector.push_back(colResults);
	}
	delete[] corrMat_host;
#endif

	
	czas = (stop - start);// / (double)1000;
	cout << "Czas obliczen w cuda = " << czas << "ms. " << endl;

	cudaDrawDirectionHeatMap(outpVector);
	
	//VisualizeCC vcc;
	//vcc.drawDirectionHeatMap(outpVector);
}	



void CudaInterface::cudaDrawDirectionHeatMap(vector<vector<OutpStr>>data)
{
	int height = data.size();
	int width = data[0].size();
	Mat hsv(height, width, CV_8UC3);

	Mat correlate(height, width, CV_8U);
	Mat distance(height, width, CV_8U);

	cvtColor(hsv, hsv, CV_RGB2HSV);

	int max_dist = sqrt( 2*pow(SUB_IMG - PATCH_SIZE - 1, 2));

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
		OutpStr temp = data[i][j];

		int dx = temp.m_point2.x - temp.m_point1.x;
		int dy = temp.m_point2.y - temp.m_point1.y;
		
		float dist = sqrtf(pow(dx, 2) + pow(dy, 2));
		float corr = (1 + temp.m_CCcoeff) / 2;

	//	cout << "x_dist = " << dx << " y_dist " << dy << " dist = " << dist << endl;

		int angle = (int)( (float)atan2(dy, dx) * 180 / 3.14);

		if (angle < 0)
			angle = 360 + angle;
	
		distance.at<uchar>(i, j) = 255-(int)(dist * 255.0 / (float)max_dist);
		correlate.at<uchar>(i, j) = (int)(corr * 255);


		//cout << "dist = " << dist << " S value = " << (int)(dist * 255 / max_dist) << endl;
		//cin.get();
		Vec3b tempHSV;
		tempHSV.val[0] = angle / 2;// (255 * (angle)) / 360;
		tempHSV.val[1] = (int)(dist * 255 / max_dist);
		tempHSV.val[2] = (int)(corr * 255);;

		hsv.at<Vec3b>(i, j) = tempHSV;
		}
	cvtColor(hsv, hsv, CV_HSV2BGR);
	imwrite("img/CudaVisualizeDirection.jpg", hsv);

	Mat distColor = Mat::zeros(height, width, CV_32F);
	Mat corrColor = Mat::zeros(height, width, CV_32F);

	applyColorMap(distance, distColor, COLORMAP_OCEAN);
	applyColorMap(correlate, corrColor, COLORMAP_OCEAN);
	imwrite("img/distCOLORMAP_OCEAN.jpg", distColor);
	imwrite("img/corrCOLORMAP_OCEAN.jpg", corrColor);



#if 0
	cout << "For 359 = " << (255 * (359)) / 360 << endl;
	cout << "For 90 = " << (255 * (90)) / 360 << endl;
	cout << "For 180 = " << (255 * (180)) / 360 << endl;
	cout << "For 270 = " << (255 * (270)) / 360 << endl;




	Mat test_hsv(height, width, CV_8UC3);
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
		Vec3b tempHSV;
		tempHSV.val[0] = 22.5;
		tempHSV.val[1] = 255;
		tempHSV.val[2] = 255;

		test_hsv.at<Vec3b>(i, j) = tempHSV;
		}
	cvtColor(test_hsv, test_hsv, CV_HSV2BGR);
	imwrite("img/testHSV_45.jpg", test_hsv);

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
		Vec3b tempHSV;
		tempHSV.val[0] = 60;
		tempHSV.val[1] = 255;
		tempHSV.val[2] = 255;

		test_hsv.at<Vec3b>(i, j) = tempHSV;
		}
	cvtColor(test_hsv, test_hsv, CV_HSV2BGR);
	imwrite("img/testHSV_120.jpg", test_hsv);

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
		Vec3b tempHSV;
		tempHSV.val[0] = 45;
		tempHSV.val[1] = 255;
		tempHSV.val[2] = 255;
		test_hsv.at<Vec3b>(i, j) = tempHSV;
		}
	cvtColor(test_hsv, test_hsv, CV_HSV2BGR);
	imwrite("img/testHSV_90.jpg", test_hsv);

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
		Vec3b tempHSV;
		tempHSV.val[0] = 90;
		tempHSV.val[1] = 255;
		tempHSV.val[2] = 255;
		test_hsv.at<Vec3b>(i, j) = tempHSV;
		}
	cvtColor(test_hsv, test_hsv, CV_HSV2BGR);
	imwrite("img/testHSV_180.jpg", test_hsv);
#endif
}


void CudaInterface::deviceInfo(){
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	printf("There are %d CUDA devices.\n", devCount);

	hipDeviceProp_t devProp;
	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&devProp, i);
		printf("Major revision number:         %d\n", devProp.major);
		printf("Minor revision number:         %d\n", devProp.minor);
		printf("Name:                          %s\n", devProp.name);
		printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
		printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", devProp.regsPerBlock);
		printf("Warp size:                     %d\n", devProp.warpSize);
		printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
		printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
		printf("Clock rate:                    %d\n", devProp.clockRate);
		printf("Total constant memory:         %lu\n", devProp.totalConstMem);
		printf("Texture alignment:             %lu\n", devProp.textureAlignment);
		printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
		printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
		printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	}
}

